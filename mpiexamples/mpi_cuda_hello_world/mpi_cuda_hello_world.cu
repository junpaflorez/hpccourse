#include <stdio.h>
#include "someDefinitions.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>


void cudaCall(int rank){
    FILE *fp;
    int device;
    hipGetDevice(&device);
    char buffer[6];
    sprintf(buffer,"file%d",rank);
    fp = fopen(buffer,"w");
    fprintf(fp,"Number of device is %d\n",device);
}
