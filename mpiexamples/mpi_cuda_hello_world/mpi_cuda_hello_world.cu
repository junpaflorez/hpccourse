#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "someDefinitions.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>

// Error handling macro
#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        printf("CUDA error calling, code is %d\n", err);}



__global__ void test (float *number, float *res){
    res[0] = number[0] * number[0];
}

void cudaCall(int rank, float *number){
    FILE *fp;
    hipDeviceProp_t prop;
    int device;

    CUDA_CHECK(hipGetDevice(&device));

    hipGetDeviceProperties(&prop, device);

    char buffer[6];
    sprintf(buffer,"file%d",rank);
    fp = fopen(buffer,"w");
    fprintf(fp,"Number of device is %d and the name is %s\n",device, prop.name);
    fclose(fp);


    float *h_res = NULL;
    h_res = (float*)malloc(1*sizeof(float));
    float *d_res = NULL;
    hipMalloc((void**)&d_res, 1*sizeof(float));

    float *d_number = NULL;
    CUDA_CHECK(hipMalloc((void**)&d_number, 1*sizeof(float)));

    hipMemcpy(d_number,number, 1*sizeof(float), hipMemcpyHostToDevice);

    test<<<1,1>>>(d_number, d_res);

    hipMemcpy(h_res, d_res, 1*sizeof(float), hipMemcpyDeviceToHost);

    number[0] = h_res[0];

    hipFree(d_res);
    free(h_res);
}
